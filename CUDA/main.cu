#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/*
*Error handlig function
*/
static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
	printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
	exit( EXIT_FAILURE );
	}
}



int main(int argc, char** argv)
{
hipDeviceProp_t prop; //vlastnosti gpu
int deviceCount; //device

HANDLE_ERROR(hipGetDeviceCount(&deviceCount));
printf("%d GPU CUDA device(s) found\n", deviceCount);
return 0;
}
