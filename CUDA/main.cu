#include <stdio.h>
#include <stdlib.h>
#include "eot.cuh" //eot sort in cuda
#include "constant.h"

void testSorted(int a[],int size)
{
	bool sorted = true;
	printf("Testuji serazenou posloupnost --->\n");
	for(int i=1;i<size;i++){
		if(a[i-1] > a[i]){
			sorted = false;
			break;
		}
	}
	if(sorted){
		printf("Serazeno\n");
	}else{
		printf("Neserazeno --->\n");
	}
}

void generateArray(int a[],int size)
{
	srand(time(NULL));
	for(int i=0;i<size;i++){
	  a[i]=(rand() % MODULE);
	}
}


void printArray(int a[],int size)
{
	for(int i=0;i<size;i++){
		printf("%d ",a[i]);
	}
	printf("\n");
}

int main(int argc, char** argv)
{
int* a = new int[ARRAY_SIZE];

generateArray(a, ARRAY_SIZE);

#ifdef DEBUG_GLOBAL
printArray(a,ARRAY_SIZE);
#endif

oddeven(a,ARRAY_SIZE);
testSorted(a,ARRAY_SIZE);

#ifdef DEBUG_GLOBAL
printArray(a,ARRAY_SIZE);
#endif

return 0;
}
