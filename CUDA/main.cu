#include <stdio.h>
#include <stdlib.h>
#include "eot.cuh" //eot sort in cuda
#include "constant.h"

void generateArray(int a[],int size)
{
	srand(time(NULL));
	for(int i=0;i<size;i++){
	  a[i]=(rand() % MODULE);
	}
}


void printArray(int a[],int size)
{
	for(int i=0;i<size;i++){
		printf("%d ",a[i]);
	}
	printf("\n");
}

int main(int argc, char** argv)
{
int* a = new int[ARRAY_SIZE];

generateArray(a, ARRAY_SIZE);
printArray(a,ARRAY_SIZE);

oddeven(a,ARRAY_SIZE);

printArray(a,ARRAY_SIZE);
return 0;
}
