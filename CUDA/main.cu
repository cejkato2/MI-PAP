#include "hip/hip_runtime.h"
// Utilities and system includes
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

#define ARRAY_SIZE 8
#define MODULE 100

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/*
*Error handlign function
*/
static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
	printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
	exit( EXIT_FAILURE );
	}
}


// compare and swap; copies from the f to t, swapping f[i] and
// f[j] if the higher-index value is smaller; it is required that i < j
__device__ void cas(int *f, int *t, int i, int j, int n, int me)
{
	if (i < 0 || j >= n) return;
	if (me == i) {
		if (f[i] > f[j]) t[me] = f[j];
		else t[me] = f[i];
	} else { // me == j
		if (f[i] > f[j]) t[me] = f[i];
		else t[me] = f[j];
	}
}

// does one iteration of the sort
__global__ void oekern(int *da, int *daaux, int n, int iter)
{
	int bix = blockIdx.x; // block number within grid
	if (iter % 2) {
		if (bix % 2) cas(da, daaux, bix - 1, bix, n, bix);
		else cas(da, daaux, bix, bix + 1, n, bix);
	} else {
		if (bix % 2) cas(da, daaux, bix, bix + 1, n, bix);
		else cas(da, daaux, bix - 1, bix, n, bix);
	}
}


void oddeven(int *ha, int n)
{
	int *da;
	int dasize = n * sizeof(int);

	HANDLE_ERROR(hipMalloc((void **)&da, dasize));
	HANDLE_ERROR(hipMemcpy(da, ha, dasize, hipMemcpyHostToDevice));
	// the array daaux will serve as "scratch space"
	int *daaux;
	HANDLE_ERROR(hipMalloc((void **)&daaux, dasize));
	dim3 dimGrid(n, 1);
	dim3 dimBlock(1, 1, 1);
	int *tmp;
	for (int iter = 1; iter <= n; iter++) {
		oekern <<< dimGrid, dimBlock >>> (da, daaux, n, iter);
		hipDeviceSynchronize();
		if (iter < n) {	
		// swap pointers
			tmp = da;
			da = daaux;
			daaux = tmp;
		} else{
			hipMemcpy(ha, daaux, dasize, hipMemcpyDeviceToHost); //copy results
		}
	}
	//free malocs
	HANDLE_ERROR(hipFree(da));
	HANDLE_ERROR(hipFree(daaux));
}


void generateArray(int a[],int size)
{
	for(int i=0;i<size;i++){
	  a[i]=(rand() % MODULE);
	}
}


void printArray(int a[],int size)
{
	for(int i=0;i<size;i++){
		printf("%d ",a[i]);
	}
	printf("\n");
}

int main(int argc, char** argv)
{
hipDeviceProp_t prop; //vlastnosti gpu
int whichDev; //device

int* a = new int[ARRAY_SIZE];

generateArray(a, ARRAY_SIZE);
printArray(a,ARRAY_SIZE);

//HANDLE_ERROR(hipGetDevice(&whichDev));
//HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDev));

//printf("MultiProcCount->%d\n",prop.multiProcessorCount);
return 0;
}
