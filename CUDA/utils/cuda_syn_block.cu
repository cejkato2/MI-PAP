#ifndef CUDA_SYN_BLOCK_CU_
#define CUDA_SYN_BLOCK_CU_

#include <hip/hip_runtime.h>

/*http://aggregate.org/MAGIC/#GPU SyncBlocks*/

inline __device__ void __syncblocks(volatile unsigned int *barnos) 
{ 

/* First, sync within each Block */
__syncthreads();
/* Pick a representative from each (here, 1D) block */
if (threadIdx.x == 0) {
  /* Get my barrier number */
  int barno = barnos[blockIdx.x] + 1;
  int hisbarno;
  int who = (blockIdx.x + 1) % gridDim.x;
  /* Check in at barrier */
  barnos[blockIdx.x] = barno;
  /* Scan for all here or somebody passed */
  do {
    /* Wait for who */
    do {
      hisbarno = barnos[who];
    } while (hisbarno < barno);
    /* Bump to next who */
    if (++who >= gridDim.x) who = 0;
  } while ((hisbarno == barno) && (who != blockIdx.x));
  /* Tell others we are all here */
  barnos[blockIdx.x] = barno + 1;
}
/* Rejoin with rest of my Block */
__syncthreads();

} 

#endif
