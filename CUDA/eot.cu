#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include "eot.cuh"
#include "utils.cuh"
#include "constant.h"
#include <math.h>


// compare and swap; copies from the f to t, swapping f[i] and
// f[j] if the higher-index value is smaller; it is required that i < j
__device__ void cas(int *f, int *t, int i, int j, int n, int my_global_pos, int me)
{
	if (i < 0 || j >= n) return;
	if (me == i) {
		if (f[i] > f[j]) t[me] = f[j];
		else t[me] = f[i];
	} else { // me == j
		if (f[i] > f[j]) t[me] = f[i];
		else t[me] = f[j];
	}
}

// does one iteration of the sort
__global__ void oekern(int *da, int *daaux, int n, int iter)
{
int tix=threadIdx.x;

	if( (iter%2) == 1){
	//provadej LS vymenu
	if( (tix%2) == 0){
		cas(da,daaux,tix,tix+1,n,n,tix);
		}else{
		cas(da,daaux,tix-1,tix,n,n,tix);
		}
	}else{
	//provadej SL vymenu
	if( (tix%2) == 1){
		cas(da,daaux,tix,tix+1,n,n,tix);
		}else{
		cas(da,daaux,tix-1,tix,n,n,tix);
		}
	}
//synchronizuj vlakna v bloku
	__syncthreads();
}


void oddeven(int *ha, int n)
{
	int *da;
	int dasize = n * sizeof(int);

	HANDLE_ERROR(hipMalloc((void **)&da, dasize));
	HANDLE_ERROR(hipMemcpy(da, ha, dasize, hipMemcpyHostToDevice));
	// the array daaux will serve as "scratch space"
	int *daaux;
	HANDLE_ERROR(hipMalloc((void **)&daaux, dasize));

	int numOfBlocks = (int) ceil(ARRAY_SIZE/NUM_OF_THREADS); //number of blocks	

	dim3 dimGrid(numOfBlocks, 1);
	dim3 dimBlock(NUM_OF_THREADS, 1, 1);

	for (int iter = 1; iter <= n; iter++) {
		oekern <<< dimGrid, dimBlock >>> (da, daaux, n, iter); //one iteration
		hipDeviceSynchronize();
		if (iter < n) {	
			hipMemcpy(da,daaux,dasize,hipMemcpyDeviceToDevice); //refresh copy
		} else{
			hipMemcpy(ha, daaux, dasize, hipMemcpyDeviceToHost); //copy results
		}
	}
	//free malocs
	HANDLE_ERROR(hipFree(da));
	HANDLE_ERROR(hipFree(daaux));
}

