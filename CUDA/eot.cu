#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include "eot.cuh"
#include "utils/utils.cuh"
#include "constant.h"
#include <math.h>
#include <utils/cuda_syn_block.cu> //synchronizace napric barierama -- prasarna vkladat primo ceckovy kod, ale :
//You must define __device__ functions within the compilation unit they are called in, and their behavior similar to functions declared with the c++ inline keyword.

// compare and swap; copies from the f to t, swapping f[i] and
// f[j] if the higher-index value is smaller; it is required that i < j
__device__ void cas(int *f, int *t, int i, int j, int n, int my_global_pos, int me)
{
	if (i < 0 || j >= n) return;
	if (me == i) {
		if (f[i] > f[j]) t[me] = f[j];
		else t[me] = f[i];
	} else { // me == j
		if (f[i] > f[j]) t[me] = f[i];
		else t[me] = f[j];
	}
}

// does one iteration of the sort
/*
da - ukazatel od prvniho pole
daaux - ukazatel do temp pole
n - velikost trideneho pole
iter - aktualni iterace
barnos - pomocne pole pro synchro mezi bloky
*/
__global__ void oekern(int *da, int *daaux, int n, int iter, volatile unsigned int* barnos)
{
int tix=threadIdx.x;
int d_index=blockIdx.x*NUM_OF_THREADS + tix; //globalni index v poli v hlavni pameti

__syncblocks(barnos);

	if( (iter%2) == 1){
	//provadej LS vymenu
		if( (tix%2) == 0){
			cas(da,daaux,d_index,d_index+1,n,d_index,d_index);
		}else{
			cas(da,daaux,d_index-1,d_index,n,d_index,d_index);
		}
	}else{
	//provadej SL vymenu
		if( (tix%2) == 1){
			cas(da,daaux,d_index,d_index+1,n,d_index,d_index);
		}else{
			cas(da,daaux,d_index-1,d_index,n,d_index,d_index);
		}
	}
}


void oddeven(int *ha, int n)
{
	int *da;
	int dasize = n * sizeof(int);

	HANDLE_ERROR(hipMalloc((void **)&da, dasize));
	HANDLE_ERROR(hipMemcpy(da, ha, dasize, hipMemcpyHostToDevice));
	// the array daaux will serve as "scratch space"
	int *daaux;
	HANDLE_ERROR(hipMalloc((void **)&daaux, dasize));

	int numOfBlocks = (int) ceil(ARRAY_SIZE/NUM_OF_THREADS); //number of blocks
	
	// ===== alokuj pole pro synchro =====
	unsigned int* h_barnos = new unsigned int[numOfBlocks]; //pomocne pole
	//inicializuj pole cislem 0
	for(int i=0;i<numOfBlocks;i++)	h_barnos[i] = 0;
	
	volatile unsigned int* barnos; //nase promenna, kterou budem ladovat cudu
	HANDLE_ERROR(hipMalloc((void **) &barnos, sizeof(int)*numOfBlocks)); //alokujem si na to mistecko
	HANDLE_ERROR(hipMemcpy((void*)barnos,h_barnos, sizeof(int)*numOfBlocks,hipMemcpyHostToDevice)); //naládujem pole
	
	delete[] h_barnos; //a uklidime po sobe	pole,co nepotrebujeme				

	// ===== priprav sturkturu pro deleni problemu =====
	dim3 dimGrid(numOfBlocks, 1);
	dim3 dimBlock(NUM_OF_THREADS, 1, 1);

	//deme na problem	
	for (int iter = 1; iter <= n; iter++) {
		oekern <<< dimGrid, dimBlock >>> (da, daaux, n, iter,barnos); //one iteration
		hipDeviceSynchronize();
		if (iter < n) {	
			hipMemcpy(da,daaux,dasize,hipMemcpyDeviceToDevice); //refresh copy
		} else{
			hipMemcpy(ha, daaux, dasize, hipMemcpyDeviceToHost); //copy results
		}
	}
	//free malocs
	HANDLE_ERROR(hipFree(da));
	HANDLE_ERROR(hipFree(daaux));
}

