#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include "eot.cuh"
#include "utils/utils.cuh"
#include "constant.h"
#include <math.h>
#include "utils/cuda_comparer.cu" //komparator pro cudu
#include <utils/cuda_syn_block.cu> //synchronizace napric barierama -- prasarna vkladat primo ceckovy kod, ale :
//You must define __device__ functions within the compilation unit they are called in, and their behavior similar to functions declared with the c++ inline keyword.


// compare and swap; copies from the f to t, swapping f[i] and
// f[j] if the higher-index value is smaller; it is required that i < j
/*
local_f - lokalni pole odkud se berou data
local_t - lokalni pole, kam se docasne kopiruje
global_h - globalni pole pro transpozice
i,j - indexy
n - pocet prvku
my_global_pos - globalni pozice jadra
me - moje X pozice jako threadu
phase - aktualni faze sudo licha nebo licho suda
dir - smysl trideni
my_global_x - globalni x pozice
localId - id v lokanlni pameti
col_size - pocet sloupecku
blkXDim - sirka bloku v X 
----------------------------------------------------
POZOR: DATA V TETO FAZI MUSI BYT JAK V GLOBALNI TAK V LOKALNI CASTI KOHERENTNI!! A pracuje jenom s X koordinatama vlakna v gridu
*/
__device__ void cas_row(int *local_f, int *local_t, int* global_h, 
			int i, int j, int n, int me, int phase,int dir, 
			int my_global_x, int my_global_pos, int localId,int col_size,int blkXDim)
{
//1) Index I mimo blok AND moje globalni_pozice = krajni prvky N
	if((my_global_x == 0) && i < 0){  //kontrola leve zarazky - jsi na krajni pozici a presahujes blok?
		return ;
	} 

	if((my_global_x == (col_size-1)) && j>=blkXDim){ //kontrola prave zarazky -> jsi na globalni pozici a presahujes blok?
		return ;
	}
	int ii;
	int jj; //promenne pro prepocet na faze v lokalni pameti ==> nutne 
	register bool res; //vysledek komparace
//2) Jsme v ramci globalniho pole N, muzem zacit provadet vymeny - jsme v ramci bloku mimo krajni prvky? -> pokud ano, trid ve sdilene pameti
	if( ((i>0 && i<(blkXDim-1)) && (phase == LS)) //pokud nejsi mimo v LS fazi, tak trid v ramci lokalniho pole
		   || phase == SL){ //pokud mas SL fazi, tak je vse OK a muzes vse tridit v ramci lokalniho pole
		if (me == i) { //v teto casti jsme v ramci indexu sdileneho pole
			ii=localId;
			jj=localId+1; 
			compare_k(local_f[ii], local_f[jj], dir, &res);	
			if (res == false) local_t[ii] = local_f[jj];
			else local_t[ii] = local_f[ii];
		} else { // me == j
			ii=localId-1;
			jj=localId;
			compare_k(local_f[ii], local_f[jj], dir, &res);	
			if (res == false) local_t[jj] = local_f[ii];
			else local_t[jj] = local_f[jj];
		}			
	}else{ //jinak musis komunikovat do globalni pameti, protoze jsi vlakno s krajnim indexem a 
		if(me == (blkXDim - 1)){ //pokud je me cislo rovno  krajnimu cislu vlakna (tj NUM_OF_THREADS -1) -> jsem i
			ii=localId;
			compare_k(local_f[ii], global_h[my_global_pos+1], dir, &res);
			if(res == false) local_t[ii] = global_h[my_global_pos+1]; //pokud jsem vetsi jak muj glob. soused, tak si upravim v lokalni pameti data
			else local_t[ii] = local_f[ii]; //jinak jsem na to spravne a prekopiruju si to do tmp pole
		}else{//jsem j
			jj=localId;
			compare_k(global_h[my_global_pos-1], local_f[jj], dir, &res);
			if(res == false) local_t[jj] = global_h[my_global_pos-1]; //pokud je muj globalni soused vetsi jak ja, tak si uravim v lokalni pameti
			else local_t[jj] = local_f[jj]; //jinak jsem na tom dobre a nic menit nemusim
		}	
	}
}

// does one iteration of the sort
/*
da - ukazatel od prvniho pole
daaux - ukazatel do temp pole
n - velikost trideneho pole
iter - aktualni iterace
barnos - pomocne pole pro synchro mezi bloky
*/
__global__ void oekern(int *h_da, volatile unsigned int* barnos, int row_size, int col_size,int blkDimX)
{
//vypocet souradnice X a Y
	int tix=threadIdx.x;
	int tiy=threadIdx.y;
	int localId=tiy*blkDimX + tix;
	
	//globalni index do pole	
	int d_index=tiy*col_size + blkDimX*blockIdx.x+tix; //globalni index v poli v hlavni pameti
	
	//souradnice x,y globalniho pole
	int x = blkDimX*blockIdx.x+tix;
	int y = tiy;

	//1) Kazde vlakno nakopiruje do lokalni pameti bloku sve data
	__shared__ int sData[NUM_OF_THREADS]; //alokace lokalni pameti
	__shared__ int sData_aux[NUM_OF_THREADS]; //temp datove pole
	sData[localId] = h_da[d_index]; //prekopiruji si data do lokalni pameti

	////////// DEBUG ///////////
// 	h_da[d_index] = d_index;	
//	h_da[d_index] = x;
//	h_da[d_index] = y;

//2) Pockame, az to udelaji vsichni ve vsech blocich
	__syncblocks(barnos); 

//3) N-krat budeme opakovat transpozice nad svou casti dat
//Pozn. : SL liche jsou v ramci sdilene pameti. U LS musi krajni vlakna komunikovat prez globalni pamet.
	unsigned int iter;
	unsigned int phase;
	int dir=ASCENDIG; //toto se po case nahradi!

	for(iter=0; iter < col_size ; iter++){ 

		//urci fazi -> podle tveho radku
		if((iter%2) == 0){
			phase = SL;
		}else{
			phase = LS;
		}

		//urci smer razeni
		if((y%2) == 0){
			dir=ASCENDIG;
		}else{
			dir=DESCENDING;
		}		

		if(phase == SL){
		//provadej SL vymenu
			if( (tix%2) == 0){
				cas_row(sData, sData_aux, h_da, tix, tix+1, col_size, tix, phase, dir, x, d_index,localId, col_size, blkDimX);
			}else{
				cas_row(sData, sData_aux, h_da, tix-1, tix, col_size, tix, phase, dir, x, d_index,localId, col_size, blkDimX);
			}
		}else{
		//provadej LS vymenu => zacina se zde v prvni iteraci
			if( (tix%2) == 1){
				cas_row(sData, sData_aux, h_da, tix,tix+1, col_size, tix, phase, dir, x, d_index,localId, col_size, blkDimX);
			}else{
				cas_row(sData, sData_aux, h_da, tix-1, tix, col_size, tix, phase, dir, x, d_index,localId, col_size, blkDimX);
			}
		}
//4) Dokoncili jsme jednu vymenu, pockame na vsechny bloky a krajni vlakna osvezi data na svojich pozicich v globalni pameti
	__syncblocks(barnos); //pockame az vsichni dodelaji krok
	
	//pouze krajni reprezentanti udelaji atualizace v globalni pameti
	if(tix==0 || tix==(NUM_OF_THREADS-1)){
		h_da[d_index] = sData_aux[localId]; 
	}

	sData[localId]=sData_aux[localId]; //kazde vlakno si navic osvezi sva data z temp pole

	//prubezne kopirovani do globalni pamet
	#ifdef DEBUG_GLOBAL	
	h_da[d_index] = sData[localId];	
	#endif	

	__syncblocks(barnos); //a pokracovaudaMemcpy(da,daaux,dasize,hipMemcpyDeviceToDevice);t budeme, az toto dokoci vsechny vlakna ve vsech blocich	
	}
//5) Ukonceno N iteraci ---> nakopirujeme data do globalni pameti
	h_da[d_index] = sData[localId]; 
}


void ShearOddeven(int *ha, int row_count,int col_count)
{
	int *da;
	int n = row_count*col_count;
	int dasize = n * sizeof(int);

	//choose best device
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	if (num_devices > 1) {
      	int max_multiprocessors = 0, max_device = 0;
      		for (device = 0; device < num_devices; device++) {
              		hipDeviceProp_t properties;
              		hipGetDeviceProperties(&properties, device);
              		
			if (max_multiprocessors < properties.multiProcessorCount) {
                      		max_multiprocessors = properties.multiProcessorCount;
                	     	max_device = device;
        	      	}
      		}
	
      		hipSetDevice(max_device);
		
		//vystup vlastnosti
		printf("Device number --> %d\n", num_devices);
		printf("Multiprocesor count --> %d\n", max_multiprocessors);
		printf("========================================\n");
		printf("Vybrane zarizeni --> %d\n", device);
	}
	printf("Pocet threadu --> %d\n\n",NUM_OF_THREADS);

	HANDLE_ERROR(hipMalloc((void **)&da, dasize));
	HANDLE_ERROR(hipMemcpy(da, ha, dasize, hipMemcpyHostToDevice));
	// the array daaux will serve as "scratch space"
	
	int xBlkDim = NUM_OF_THREADS/row_count; //x rozmer bloku
	int yBlkDim = row_count;
	int xMatrix = col_count; //x rozmer matice	
	int numOfBlocks = (int) ceil(xMatrix/xBlkDim); //pocet bloku
	
	// ===== alokuj pole pro synchro =====
	unsigned int* h_barnos = new unsigned int[numOfBlocks]; //pomocne pole
	//inicializuj pole cislem 0
	for(int i=0;i<numOfBlocks;i++)	h_barnos[i] = 0;
	
	volatile unsigned int* barnos; //nase promenna, kterou budem ladovat cudu
	HANDLE_ERROR(hipMalloc((void **) &barnos, sizeof(int)*numOfBlocks)); //alokujem si na to mistecko
	HANDLE_ERROR(hipMemcpy((void*)barnos,h_barnos, sizeof(int)*numOfBlocks,hipMemcpyHostToDevice)); //naládujem pole
	
	delete[] h_barnos; //a uklidime po sobe	pole,co nepotrebujeme				

	// ===== priprav sturkturu pro deleni problemu =====
	dim3 dimGrid(numOfBlocks, 1); //pustime to na pocet bloku (jak jsme spocitali)
	dim3 dimBlock(xBlkDim,yBlkDim, 1); //a kazdy blok bude mit rozmery

	// ===== deme na problem =====	
	oekern <<< dimGrid, dimBlock >>> (da, barnos, row_count, col_count,xBlkDim); //eot sort v radku
	hipDeviceSynchronize();

	HANDLE_ERROR(hipMemcpy(ha,da,dasize,hipMemcpyDeviceToHost));
	
	//free malocs
	HANDLE_ERROR(hipFree(da));
	HANDLE_ERROR(hipFree((void *) barnos));
}

