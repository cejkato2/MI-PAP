#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include "eot.cuh"
#include "utils/utils.cuh"
#include "constant.h"
#include <math.h>
#include "utils/cuda_comparer.cu" //komparator pro cudu
#include <utils/cuda_syn_block.cu> //synchronizace napric barierama -- prasarna vkladat primo ceckovy kod, ale :
//You must define __device__ functions within the compilation unit they are called in, and their behavior similar to functions declared with the c++ inline keyword.


// compare and swap; copies from the f to t, swapping f[i] and
// f[j] if the higher-index value is smaller; it is required that i < j
/*
local_f - lokalni pole odkud se berou data
local_t - lokalni pole, kam se docasne kopiruje
global_h - globalni pole pro transpozice
i,j - indexy
n - pocet prvku
my_global_pos - globalni pozice jadra
me - moje X pozice jako threadu
phase - aktualni faze sudo licha nebo licho suda
dir - smysl trideni
my_global_x - globalni x pozice
localId - id v lokanlni pameti
col_size - pocet sloupecku
blkXDim - sirka bloku v X 
----------------------------------------------------
POZOR: DATA V TETO FAZI MUSI BYT JAK V GLOBALNI TAK V LOKALNI CASTI KOHERENTNI!! A pracuje jenom s X koordinatama vlakna v gridu
*/
__device__ void cas_row(int *local_f, int *local_t, int* global_h, 
			int i, int j, int n, int me, int phase,int dir, 
			int my_global_x, int my_global_pos, int localId,int col_size,int blkXDim)
{
//1) Index I mimo blok AND moje globalni_pozice = krajni prvky N
	if((my_global_x == 0) && i < 0){  //kontrola leve zarazky - jsi na krajni pozici a presahujes blok?
		return ;
	} 

	if((my_global_x == (col_size-1)) && j>=blkXDim){ //kontrola prave zarazky -> jsi na globalni pozici a presahujes blok?
		return ;
	}
	int ii;
	int jj; //promenne pro prepocet na faze v lokalni pameti ==> nutne 
	register bool res; //vysledek komparace
//2) Jsme v ramci globalniho pole N, muzem zacit provadet vymeny - jsme v ramci bloku mimo krajni prvky? -> pokud ano, trid ve sdilene pameti
	if( ((i>0 && i<(blkXDim-1)) && (phase == LS)) //pokud nejsi mimo v LS fazi, tak trid v ramci lokalniho pole
		   || phase == SL){ //pokud mas SL fazi, tak je vse OK a muzes vse tridit v ramci lokalniho pole
		if (me == i) { //v teto casti jsme v ramci indexu sdileneho pole
			ii=localId;
			jj=localId+1; 
			compare_k(local_f[ii], local_f[jj], dir, &res);	
			if (res == false) local_t[ii] = local_f[jj];
			else local_t[ii] = local_f[ii];
		} else { // me == j
			ii=localId-1;
			jj=localId;
			compare_k(local_f[ii], local_f[jj], dir, &res);	
			if (res == false) local_t[jj] = local_f[ii];
			else local_t[jj] = local_f[jj];
		}			
	}else{ //jinak musis komunikovat do globalni pameti, protoze jsi vlakno s krajnim indexem a 
		if(me == (blkXDim - 1)){ //pokud je me cislo rovno  krajnimu cislu vlakna (tj NUM_OF_THREADS -1) -> jsem i
			ii=localId;
			compare_k(local_f[ii], global_h[my_global_pos+1], dir, &res);
			if(res == false) local_t[ii] = global_h[my_global_pos+1]; //pokud jsem vetsi jak muj glob. soused, tak si upravim v lokalni pameti data
			else local_t[ii] = local_f[ii]; //jinak jsem na to spravne a prekopiruju si to do tmp pole
		}else{//jsem j
			jj=localId;
			compare_k(global_h[my_global_pos-1], local_f[jj], dir, &res);
			if(res == false) local_t[jj] = global_h[my_global_pos-1]; //pokud je muj globalni soused vetsi jak ja, tak si uravim v lokalni pameti
			else local_t[jj] = local_f[jj]; //jinak jsem na tom dobre a nic menit nemusim
		}	
	}
}


/*
local_f,local_t - ostre a tmp pole
i,j - indexy prvku na porovnani
n - pocet prvku
meId - moje lokalni I lokalni ID
blkXDim - dimenze bloku
*/
//POZOR!! Indexy i a j jsou efektivni ukazatele do lokalni pameti
__device__ void cas_col(int *local_f, int *local_t, int i, int j, int max_index, int meId)
{
	if(i < 0 || j>=max_index){ //kontrola zarazek
		return;
	}

	if(meId == i){ 
	//me==i
		if(local_f[i] > local_f[j]){
			local_t[meId]=local_f[j];	
		}else{
			local_t[meId]=local_f[i];
		}
	}else{
	//me==j
		if(local_f[i] > local_f[j]){
			local_t[meId]=local_f[i];
		}else{
			local_t[meId]=local_f[j];
		}
	}
}

/*
da - ukazatel od prvniho pole
daaux - ukazatel do temp pole
n - velikost trideneho pole
iter - aktualni iterace
barnos - pomocne pole pro synchro mezi bloky
blkDimX - dimense X bloku
*/
__global__ void ShearOekern(int *h_da, volatile unsigned int* barnos, int row_size, int col_size,int blkDimX)
{
//vypocet souradnice X a Y
	int tix=threadIdx.x;
	int tiy=threadIdx.y;
	int localId=tiy*blkDimX + tix;
	
	//globalni index do pole	
	int d_index=tiy*col_size + blkDimX*blockIdx.x+tix; //globalni index v poli v hlavni pameti
	
	//souradnice x,y globalniho pole
	int x = blkDimX*blockIdx.x+tix;
	int y = tiy;

	//1) Kazde vlakno nakopiruje do lokalni pameti bloku sve data
	__shared__ int sData[NUM_OF_THREADS]; //alokace lokalni pameti
	__shared__ int sData_aux[NUM_OF_THREADS]; //temp datove pole
	sData[localId] = h_da[d_index]; //prekopiruji si data do lokalni pameti

	int numOfPhases = 2 * ((int)floor(log2((float)row_size))) + 1; //pocet iteraci shearsortu
	////////// DEBUG ///////////
// 	h_da[d_index] = d_index;	
//	h_da[d_index] = x;
//	h_da[d_index] = y;

	for(int act_iter=0; act_iter < numOfPhases; act_iter++){
	//2) Pockame, az to udelaji vsichni ve vsech blocich
		__syncblocks(barnos); 

		//urceni faze --> podle aktualni iterace
		int sh_iter;
		if((act_iter%2)==0){
			sh_iter=SH_ROW;
		}else{
			sh_iter=SH_COL;
		}	
	
	//3) N-krat budeme opakovat transpozice nad svou casti dat
	//Pozn. : SL liche jsou v ramci sdilene pameti. U LS musi krajni vlakna komunikovat prez globalni pamet.
		unsigned int iter;
		unsigned int phase; //SL nebo LS faze porovnani
		int dir; //smer razeni
       		int numOfIter;

		//vyber poctu iteraci	
		if(sh_iter == SH_ROW){
			numOfIter=col_size;
		}else{
			numOfIter=row_size;
		}	

		for(iter=0; iter < numOfIter ; iter++){ 
	
			//urci fazi -> podle tveho radku
			if((iter%2) == 0){
				phase = SL;
			}else{
				phase = LS;
			}
	
			//urci smer razeni
			if(sh_iter == SH_ROW){
				if((y%2) == 0){
					dir=ASCENDIG;
				}else{
					dir=DESCENDING;
				}
			}else{
				dir=ASCENDIG;
			}
					
	
			
			if(sh_iter == SH_ROW){
				if(phase == SL){
				//provadej SL vymenu
					if( (tix%2) == 0){
						cas_row(sData, sData_aux, h_da, tix, tix+1, col_size, tix, phase, dir, x, d_index,localId, col_size, blkDimX);
					}else{
						cas_row(sData, sData_aux, h_da, tix-1, tix, col_size, tix, phase, dir, x, d_index,localId, col_size, blkDimX);
					}
				}else{
				//provadej LS vymenu => zacina se zde v prvni iteraci
					if( (tix%2) == 1){
						cas_row(sData, sData_aux, h_da, tix,tix+1, col_size, tix, phase, dir, x, d_index,localId, col_size, blkDimX);
					}else{
						cas_row(sData, sData_aux, h_da, tix-1, tix, col_size, tix, phase, dir, x, d_index,localId, col_size, blkDimX);
					}
				}
			}else{
			//SH_COLUMN faze
				if(phase == SL){
					if((tiy%2) == 0){
						cas_col(sData, sData_aux, localId, localId+blkDimX, NUM_OF_THREADS, localId);	
					}else{
						cas_col(sData, sData_aux, localId-blkDimX, localId, NUM_OF_THREADS, localId);
					}
				}else{
					if((tiy%2) == 1){
						cas_col(sData, sData_aux, localId, localId+blkDimX, NUM_OF_THREADS, localId);
					}else{
						cas_col(sData, sData_aux, localId-blkDimX, localId, NUM_OF_THREADS, localId);
					}
				}
			}
	//4) Do	koncili jsme jednu vymenu, pockame na vsechny bloky a krajni vlakna osvezi data na svojich pozicich v globalni pameti
		__syncblocks(barnos); //pockame az vsichni dodelaji krok
		
		//pouze krajni reprezentanti udelaji atualizace v globalni pameti
		if(tix==0 || tix==(NUM_OF_THREADS-1)){
			h_da[d_index] = sData_aux[localId]; 
		}
	
		sData[localId]=sData_aux[localId]; //kazde vlakno si navic osvezi sva data z temp pole

		//prubezne kopirovani do globalni pamet
		#ifdef DEBUG_GLOBAL	
		h_da[d_index] = sData[localId];	
		#endif	
	
		__syncblocks(barnos); //a pokracovaudaMemcpy(da,daaux,dasize,hipMemcpyDeviceToDevice);t budeme, az toto dokoci vsechny vlakna ve vsech blocich	
		}
	}
//5) Ukonceno N iteraci ---> nakopirujeme data do globalni pameti
	h_da[d_index] = sData[localId]; 
}


void ShearOddeven(int *ha, int row_count,int col_count)
{
	int *da;
	int n = row_count*col_count;
	int dasize = n * sizeof(int);
	//mereni casu
	hipEvent_t start, stop;
	float elapsedTime;
	
	HANDLE_ERROR( hipEventCreate( &start ) );
	HANDLE_ERROR( hipEventCreate( &stop ) );


	//choose best device
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	if (num_devices > 1) {
      	int max_multiprocessors = 0, max_device = 0;
      		for (device = 0; device < num_devices; device++) {
              		hipDeviceProp_t properties;
              		hipGetDeviceProperties(&properties, device);
              		
			if (max_multiprocessors < properties.multiProcessorCount) {
                      		max_multiprocessors = properties.multiProcessorCount;
                	     	max_device = device;
        	      	}
      		}
	
      		hipSetDevice(max_device);
		
		//vystup vlastnosti
		printf("Device number --> %d\n", num_devices);
		printf("Multiprocesor count --> %d\n", max_multiprocessors);
		printf("========================================\n");
		printf("Vybrane zarizeni --> %d\n", device);
	}
	printf("Pocet threadu --> %d\n\n",NUM_OF_THREADS);

	HANDLE_ERROR(hipMalloc((void **)&da, dasize));
	HANDLE_ERROR(hipMemcpy(da, ha, dasize, hipMemcpyHostToDevice));
	// the array daaux will serve as "scratch space"
	
	int xBlkDim = NUM_OF_THREADS/row_count; //x rozmer bloku
	int yBlkDim = row_count;
	int xMatrix = col_count; //x rozmer matice	
	int numOfBlocks = (int) ceil(xMatrix/xBlkDim); //pocet bloku
	
	// ===== alokuj pole pro synchro =====
	unsigned int* h_barnos = new unsigned int[numOfBlocks]; //pomocne pole
	//inicializuj pole cislem 0
	for(int i=0;i<numOfBlocks;i++)	h_barnos[i] = 0;
	
	volatile unsigned int* barnos; //nase promenna, kterou budem ladovat cudu
	HANDLE_ERROR(hipMalloc((void **) &barnos, sizeof(int)*numOfBlocks)); //alokujem si na to mistecko
	HANDLE_ERROR(hipMemcpy((void*)barnos,h_barnos, sizeof(int)*numOfBlocks,hipMemcpyHostToDevice)); //naládujem pole
	
	delete[] h_barnos; //a uklidime po sobe	pole,co nepotrebujeme				

	// ===== priprav sturkturu pro deleni problemu =====
	dim3 dimGrid(numOfBlocks, 1); //pustime to na pocet bloku (jak jsme spocitali)
	dim3 dimBlock(xBlkDim,yBlkDim, 1); //a kazdy blok bude mit rozmery

	printf("Pocet bloku-->%d\n",numOfBlocks);
	printf("Rozmery bloku--> %d radku, %d sloupcu\n",yBlkDim,xBlkDim);
	// ===== deme na problem =====	
	HANDLE_ERROR( hipEventRecord( start, 0 ) );
	ShearOekern <<< dimGrid, dimBlock >>> (da, barnos, row_count, col_count,xBlkDim); //shearsort
	hipDeviceSynchronize();
	HANDLE_ERROR(hipMemcpy(ha,da,dasize,hipMemcpyDeviceToHost));
	
	HANDLE_ERROR( hipEventRecord( stop, 0 ) );

	//zjisteni casu
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf("\n\n-----------------------------------------------------\n");
	printf( "GPU čas: %g ms\n", elapsedTime );
	printf("-----------------------------------------------------\n\n");

	//uklizeni
	HANDLE_ERROR( hipEventDestroy( start ) );
	HANDLE_ERROR( hipEventDestroy( stop ) );

	//free malocs
	HANDLE_ERROR(hipFree(da));
	HANDLE_ERROR(hipFree((void *) barnos));
}

