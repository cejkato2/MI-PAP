#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include "eot.cuh"
#include "utils/utils.cuh"
#include "constant.h"
#include <math.h>
#include <utils/cuda_syn_block.cu> //synchronizace napric barierama -- prasarna vkladat primo ceckovy kod, ale :
//You must define __device__ functions within the compilation unit they are called in, and their behavior similar to functions declared with the c++ inline keyword.

// compare and swap; copies from the f to t, swapping f[i] and
// f[j] if the higher-index value is smaller; it is required that i < j
/*
local_f - lokalni pole odkud se berou data
local_t - lokalni pole, kam se docasne kopiruje
global_h - globalni pole pro transpozice
i,j - indexy
n - pocet prvku
my_global_pos - globalni pozice jadra
me - moje cislo jadra
phase - aktualni faze sudo licha nebo licho suda
----------------------------------------------------
POZOR: DATA V TETO FAZI MUSI BYT JAK V GLOBALNI TAK V LOKALNI CASTI KOHERENTNI!!
*/
__device__ void cas(int *local_f, int *local_t, int* global_h, int i, int j, int n, int my_global_pos, int me, int phase)
{
//1) Index I mimo blok AND moje globalni_pozice = krajni prvky N
	if(my_global_pos == 0 && i < 0){return ;} //kontrola leve zarazky
	if(my_global_pos == (n-1) && j>=n){return ;} //kontrola prave zarazky
//2) Jsme v ramci globalniho pole N, muzem zacit provadet vymeny - jsme v ramci bloku mimo krajni prvky? -> pokud ano, trid ve sdilene pameti
	if( ((i>0 && i<(NUM_OF_THREADS-1)) && (phase == LS)) //pokud nejsi mimo v LS fazi, tak trid v ramci lokalniho pole
		   || phase == SL){ //pokud mas SL fazi, tak je vse OK a muzes vse tridit v ramci lokalniho pole
		if (me == i) { //v teto casti jsme v ramci indexu sdileneho pole 
			if (local_f[i] > local_f[j]) local_t[me] = local_f[j];
			else local_t[me] = local_f[i];
		} else { // me == j
			if (local_f[i] > local_f[j]) local_t[me] = local_f[i];
			else local_t[me] = local_f[j];
		}			
	}else{ //jinak musis komunikovat do globalni pameti, protoze jsi vlakno s krajnim indexem a 
		if(me == (NUM_OF_THREADS - 1)){ //pokud je me cislo rovno  krajnimu cislu vlakna (tj NUM_OF_THREADS -1) -> jsem i 
			if(local_f[i] > global_h[my_global_pos+1]) local_t[me] = global_h[my_global_pos+1]; //pokud jsem vetsi jak muj glob. soused, tak si upravim v lokalni pameti data
			else local_t[me] = local_f[i]; //jinak jsem na to spravne a prekopiruju si to do tmp pole
		}else{//jsem j
			if(global_h[my_global_pos-1] > local_f[j]) local_t[me] = global_h[my_global_pos-1]; //pokud je muj globalni soused vetsi jak ja, tak si uravim v lokalni pameti
			else local_t[me] = local_f[j]; //jinak jsem na tom dobre a nic menit nemusim
		}	
	}
}

// does one iteration of the sort
/*
da - ukazatel od prvniho pole
daaux - ukazatel do temp pole
n - velikost trideneho pole
iter - aktualni iterace
barnos - pomocne pole pro synchro mezi bloky
*/
__global__ void oekern(int *h_da, int n, volatile unsigned int* barnos)
{
	int tix=threadIdx.x;
	int d_index=blockIdx.x*NUM_OF_THREADS + tix; //globalni index v poli v hlavni pameti
//1) Kazde vlakno nakopiruje do lokalni pameti bloku sve data
	__shared__ int sData[NUM_OF_THREADS]; //alokace lokalni pameti
	__shared__ int sData_aux[NUM_OF_THREADS]; //temp datove pole
	sData[tix] = h_da[d_index]; //prekopiruji si data do lokalni pameti

//2) Pockame, az to udelaji vsichni ve vsech blocich
	__syncblocks(barnos); 

//3) N-krat budeme opakovat transpozice nad svou casti dat
//Pozn. : SL liche jsou v ramci sdilene pameti. U LS musi krajni vlakna komunikovat prez globalni pamet.
	unsigned int iter;
	unsigned int phase;
	for(iter=0; iter < n; iter++){ 

		//urci fazi
		if((iter%2) == 0){
			phase = SL;
		}else{
			phase = LS;
		}

		if(phase == SL){
		//provadej SL vymenu
			if( (tix%2) == 0){
				cas(sData, sData_aux, h_da, tix, tix+1, n, d_index, tix, phase);
			}else{
				cas(sData, sData_aux, h_da, tix-1, tix, n, d_index, tix, phase);
			}
		}else{
		//provadej LS vymenu => zacina se zde v prvni iteraci
			if( (tix%2) == 1){
				cas(sData, sData_aux, h_da, tix,tix+1, n, d_index, tix, phase);
			}else{
				cas(sData, sData_aux, h_da, tix-1, tix, n, d_index, tix, phase);
			}
		}
//4) Dokoncili jsme jednu vymenu, pockame na vsechny bloky a krajni vlakna osvezi data na svojich pozicich v globalni pameti
	__syncblocks(barnos); //pockame az vsichni dodelaji krok
	
	//pouze krajni reprezentanti udelaji atualizace v globalni pameti
	if(tix==0 || tix==(NUM_OF_THREADS-1)){
		h_da[d_index] = sData_aux[tix]; 
	}

	sData_aux[tix]=sData[tix]; //kazde vlakno si navic osvezi sva data z temp pole
	
	#ifdef DEBUG_GLOBAL
	h_da[d_index] = sData[tix];	
	#endif
	
	__syncblocks(barnos); //a pokracovat budeme, az toto dokoci vsechny vlakna ve vsech blocich	
	}
//5) Ukonceno N iteraci ---> nakopirujeme data do globalni pameti
	h_da[d_index] = sData_aux[tix]; 
}


void oddeven(int *ha, int n)
{
	int *da;
	int dasize = n * sizeof(int);

	HANDLE_ERROR(hipMalloc((void **)&da, dasize));
	HANDLE_ERROR(hipMemcpy(da, ha, dasize, hipMemcpyHostToDevice));
	// the array daaux will serve as "scratch space"
	int *daaux;
	HANDLE_ERROR(hipMalloc((void **)&daaux, dasize));

	int numOfBlocks = (int) ceil(ARRAY_SIZE/NUM_OF_THREADS); //number of blocks
	
	// ===== alokuj pole pro synchro =====
	unsigned int* h_barnos = new unsigned int[numOfBlocks]; //pomocne pole
	//inicializuj pole cislem 0
	for(int i=0;i<numOfBlocks;i++)	h_barnos[i] = 0;
	
	volatile unsigned int* barnos; //nase promenna, kterou budem ladovat cudu
	HANDLE_ERROR(hipMalloc((void **) &barnos, sizeof(int)*numOfBlocks)); //alokujem si na to mistecko
	HANDLE_ERROR(hipMemcpy((void*)barnos,h_barnos, sizeof(int)*numOfBlocks,hipMemcpyHostToDevice)); //naládujem pole
	
	delete[] h_barnos; //a uklidime po sobe	pole,co nepotrebujeme				

	// ===== priprav sturkturu pro deleni problemu =====
	dim3 dimGrid(numOfBlocks, 1);
	dim3 dimBlock(NUM_OF_THREADS, 1, 1);

	// ===== deme na problem =====	
	oekern <<< dimGrid, dimBlock >>> (da, n, barnos); //eot sort v radku
	
	//free malocs
	HANDLE_ERROR(hipFree(da));
	HANDLE_ERROR(hipFree(daaux));
}

