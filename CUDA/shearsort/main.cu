#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "eot.cuh" //eot sort in cuda
#include "constant.h"
#include <math.h>

//jednorozmerne pole se bere jako 2D pole
void testSorted(int a[],int size)
{
	bool sorted = true;
	printf("Testuji serazenou posloupnost ---> ");
	for(int i=1;i<size/2;i++){
		if(a[i-1] > a[i]){
			sorted = false;
			break;
		}
	}
	if(sorted){
		printf("Serazeno\n");
	}else{
		printf("Neserazeno --->\n");
	}
}

void generateArray(int a[],int size)
{
	printf("Generuji testovaci data .. (%d)\n",size);
	srand(time(NULL));
	for(int i=0;i<size;i++){
	  a[i]=(rand() % MODULE);
	}
}


void printArray(int a[],int row,int col)
{
	int index;
	for(int i=0;i<row;i++){
		for(int j=0;j<col;j++){
			index=i*col + j;		
			printf("%d ",a[index]);
		}
	printf("\n");
	}
	printf("\n");
}

int main(int argc, char** argv)
{

int ARRAY_SIZE = 16;

for(int tmp=4;tmp<26;ARRAY_SIZE*=2,tmp++){

int* a = new int[ARRAY_SIZE];

generateArray(a, ARRAY_SIZE);

//pocet radku a sloupcu problemu
int col;
int row; 


if(ARRAY_SIZE <= NUM_OF_THREADS){
	//nema cenu delit problem
	row=1;
	col=ARRAY_SIZE;
}else{
	//rozdelime problem 
	//int blkCount =  (int) ceil(ARRAY_SIZE/NUM_OF_THREADS);
	//najdem co mozny nejvetsi pocet radku
	//row = sqrt(blkCount);
	//while((blkCount%row)!=0){
	//	row--;
	//}		
	
	//col=(blkCount/row)*NUM_OF_THREADS;	
	row=NUM_OF_THREADS/4;
	col=ARRAY_SIZE/row;
}

printf("Matice --> %d radku, %d sloupcu.\n\n",row,col);

#ifdef DEBUG_GLOBAL
printArray(a,row,col);
#endif

ShearOddeven(a,row,col);

#ifdef DEBUG_GLOBAL
printArray(a,row,col);
#endif
}
return 0;
}
